
#include <hip/hip_runtime.h>
#define PADDING_X 1
#define PADDING_Y 1

extern "C" __global__ void step(const unsigned int* field, unsigned int* new_field) {
    const size_t x = blockIdx.x * blockDim.x + threadIdx.x + PADDING_X;
    const size_t y = blockIdx.y * blockDim.y + threadIdx.y + PADDING_Y;
    const size_t height = (gridDim.y + 2 * PADDING_Y);
    const size_t i = x * height + y ;

    unsigned int result = field[i];

    // top: left mid right
    const unsigned int a0 = (field[i - 1] >> 1) | (field[i - height - 1] << 31);
    const unsigned int a1 = field[i - 1];
    const unsigned int a2 = (field[i - 1] << 1) | (field[i + height - 1] >> 31);

    // middle: left right
    const unsigned int a3 = (field[i] >> 1) | (field[i - height] << 31);
    const unsigned int a4 = (field[i] << 1) | (field[i + height] >> 31);

    // bottom: left mid right
    const unsigned int a5 = (field[i + 1] >> 1) | (field[i - height + 1] << 31);
    const unsigned int a6 = field[i + 1];
    const unsigned int a7 = (field[i + 1] << 1) | (field[i + height + 1] >> 31);

    // stage 0
    const unsigned int ta0 = a0 ^ a1;
    const unsigned int a8 = ta0 ^ a2;
    const unsigned int b0 = (a0 & a1) | (ta0 & a2);

    const unsigned int ta3 = a3 ^ a4;
    const unsigned int a9 = ta3 ^ a5;
    const unsigned int b1 = (a3 & a4) | (ta3 & a5);

    const unsigned int aA = a6 ^ a7;
    const unsigned int b2 = a6 & a7;

    // stage 1
    const unsigned int ta8 = a8 ^ a9;
    const unsigned int aB = ta8 ^ aA;
    const unsigned int b3 = (a8 & a9) | (ta8 & aA);

    const unsigned int tb0 = b0 ^ b1;
    const unsigned int b4 = tb0 ^ b2;
    const unsigned int c0 = (b0 & b1) | (tb0 & b2);

    result |= aB;
    result &= (b3 ^ b4);
    result &= ~c0;

    new_field[i] = result;
}
